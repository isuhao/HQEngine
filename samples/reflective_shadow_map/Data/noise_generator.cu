#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hiprand/hiprand_kernel.h>

#define PI 3.141592654f

surface<void, cudaSurfaceType2D> surfaceWrite;

__global__ void kernel(hiprandState * randStates) {
	//init rand
	unsigned long long id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(1000, id, id, &randStates[id]);

	//get two uniform random floats
	float S1 = hiprand_uniform(&randStates[id]);
	float S2 = hiprand_uniform(&randStates[id]);

	//now write to output surface
	ushort4 output;
	output.x = __float2half_rn 	(S1);
	output.y = __float2half_rn(0.5 * sin(2.0 * PI * S2) + 0.5); //scale down to between [0..1]
	output.z = __float2half_rn(0.5 * cos(2.0 * PI * S2) + 0.5); //scale down to between [0..1]
	output.w = __float2half_rn(S1 * S1);

	int x = threadIdx.x;
	int y = blockIdx.x;
	surf2Dwrite(output, surfaceWrite, x * sizeof(ushort4), y);
}


extern "C" void cudaGenerateNoiseMapKernel(hipArray_t outputArray, unsigned int width, unsigned int height) {
	hipError_t err;
	hiprandState * randStates;

	//alloc hiprand states
	err = hipMalloc(&randStates, width * height * sizeof(hiprandState));

	//bind array to global surface
	err = cudaBindSurfaceToArray(surfaceWrite, outputArray);

	//call kernel
	kernel << < width, height >> > (randStates);

	//clean up
	err = hipFree(randStates);
}